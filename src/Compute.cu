#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <limits>
#include <random>
#include <thread>
#include <vector>
#include <unistd.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Compute.h"
#include "Scene.h"
#include "./BVHNode.h"
#include "./Functions.h"
#include "./Hittable.h"
#include "./Light.h"
#include "./Point.h"
#include "./Ray.h"
#include "./Sphere.h"
#include "./Vec.h"
#include "./PinholeCamera.h"

__global__ void deviceRender(unsigned int seed, Scene* s, BVHNode* box, unsigned char* pixels) {
  hiprandState_t state;

  hiprand_init(seed, blockIdx.x, 0, &state);

  float random = hiprand_uniform(&state);
  
  unsigned char rand = (unsigned char) random; 

  Ray r;
  Point col;
  
  pixels[blockIdx.x * 3 + ((s->getWidth() * 3) * blockIdx.y)]     = rand;
  pixels[blockIdx.x * 3 + ((s->getWidth() * 3) * blockIdx.y) + 1] = rand;
  pixels[blockIdx.x * 3 + ((s->getWidth() * 3) * blockIdx.y) + 2] = rand;
  
  // s.camera.getPrimaryRay()
}

unsigned char* GPUrender(const Scene s) {
  // this is probably really inefficient

  Scene scene = s;
  
  BVHNode box = BVHNode(*scene.getHittables(), 0, std::numeric_limits<float>::max());
  int size = scene.getHeight() * scene.getWidth() * 3;

  // device copies of needed structures
  unsigned char *gPixels;
  BVHNode* gBox;
  Scene* gScene;
  
  hipMalloc((void **) &gPixels, size);
  hipMalloc((void **) &gBox, sizeof(BVHNode));
  hipMalloc((void **) &gScene, sizeof(Scene));

  hipMemcpy(gScene, &s, sizeof(Scene), hipMemcpyHostToDevice);
  hipMemcpy(gBox, &box, sizeof(BVHNode), hipMemcpyHostToDevice);
  // hipMemcpy(gPixels, pixels, size, hipMemcpyHostToDevice);

  dim3 numBlocks(scene.getWidth(), scene.getHeight());
  int threadsPerBlock = scene.samples;
  
  deviceRender<<<numBlocks, threadsPerBlock>>>(time(0), gScene, gBox, gPixels);  

  unsigned char* pixels = (unsigned char*)malloc(size);
  hipMemcpy(pixels, gPixels, size, hipMemcpyDeviceToHost);

  return pixels;
}
// #include "Scene.h"
// #include "./BVHNode.h"
// #include "./Functions.h"
// #include "./Hittable.h"
// #include "./Light.h"
// #include "./Point.h"
// #include "./Ray.h"
// #include "./Sphere.h"
// #include "./Vec.h"
// #include "PinholeCamera.h"

// #include <cmath>
// #include <iostream>
// #include <limits>
// #include <random>
// #include <thread>
// #include <vector>

// Scene::Scene() {
//   width = 1000;
//   height = 800; 
//   pixels = new unsigned char[height * width * 3];
// }

// Scene::Scene(int w, int h, PinholeCamera camera, Point background) {
//   width = w;
//   height = h;
//   this->camera = camera;
//   this->background = background;
//   pixels = new unsigned char[height * width * 3];
// }

// unsigned char *Scene::render() const {
//   if (hittables.objects.empty())
//     return NULL;
//   BVHNode box = BVHNode(hittables, 0, std::numeric_limits<float>::max());
// #pragma omp for
//     for (int y = 0; y < height; y++) {
//       for (int x = 0; x < width * 3; x += 3) {
//         Ray r;
//         Point col;

//         std::random_device device;
//         std::mt19937 gen(device());
//         std::uniform_real_distribution<> realrand(0, 1);

//         for (int i = 0; i < samples; i++) {
//           camera.getPrimaryRay(float(x / 3) + realrand(device),
//                                float(y) + realrand(device), r);
//           col = add(col, Colour(r, bounces, box));
//         }
//         // R channel
//         pixels[y * (width * 3) + x] =
//             (col.x / samples >= 255) ? 255 : col.x / samples;
//         // G channel
//         pixels[y * (width * 3) + x + 1] =
//             (col.y / samples >= 255) ? 255 : col.y / samples;
//         // B channel
//         pixels[y * (width * 3) + x + 2] =
//             (col.z / samples >= 255) ? 255 : col.z / samples;
//       }
//     }
//   return pixels;
// }

// void Scene::newCamera(PinholeCamera p) { camera = p; }

// std::vector<Hittable *> Scene::getObjects() const { return hittables.objects; }

// void Scene::removeObject(int i) {
//   if (i > hittables.objects.size())
//     ;
//   else
//     hittables.objects.erase(hittables.objects.begin() + i);
// }

// /*--------------- Sphere stuff ---------------*/

// Point Scene::Colour(Ray r, int limit, BVHNode &sceneBox) const {
//   hitRecord rec;

//   // Checks all objects
//   if (sceneBox.hit(r, rec, 0, std::numeric_limits<float>::max()) && limit > 0) {
//     Ray scattered;
//     Point attenuation; // surface value of the rendering equation
//     Point emitted = rec.matPtr->emitted(
//         rec.u, rec.v, rec.p); // emitted value of the rendering equation
//     if (!rec.matPtr->scatter(r, rec, attenuation, scattered)) {
//       return emitted; // returns the emitted value if the object doesn't scatter
//     }
//     return emitted + attenuation * Colour(scattered, limit - 1, sceneBox);
//   } else
//     return background;
// }

// void Scene::addObject(Hittable *o) { hittables.objects.push_back(o); }
